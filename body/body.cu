#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>
#include <hip/hip_runtime.h>
#include <>

#include "body.hpp"

__global__ void calc_force_cuda(body* bodies, float g, float t, int N, int it)
{

		
	// global thread ID using the formula from the CUDA docs
	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	// check if this is actually a valid calcuation
	if (tid < N)
	{
		//runs Newton's formula
		float dx = bodies[it].x - bodies[tid].x;
		float dy = bodies[it].y - bodies[tid].y;

		float dist = dx * dx + dy * dy;

		float force, dist_inv;
		if ((bodies[it].r + bodies[tid].r) * (bodies[it].r + bodies[tid].r) >= dist)
		{
			force = 0;
			dist_inv = 0;
		}
		else
		{
			force = -(bodies[it].m * bodies[tid].m * g) / dist;
			dist_inv = rsqrtf(dist);  // ignore the squiggle lines, this is a CUDA inbuilt function
		}

		dx *= dist_inv;
		// no writing on the iterator output field
		// a) would not work anyway, because thousands of GPU threads will try to do it simultaneously and corrupt the result and
		// b) we get the correct result anyway, because when the iterator changes the previous iterator's values will be changed then
		//v_x[it] += dx * force * t / m[it];
		bodies[tid].v_x -= dx * force * t / bodies[tid].m;

		dy *= dist_inv;
		//v_y[it] += dy * force * t / m[it];
		bodies[tid].v_y -= dy * force * t / bodies[tid].m;
	}

}

__global__ void calc_force_cuda_full(body* bodies, float g, float t, int N)
{

	int tid_x = blockIdx.x * blockDim.x + threadIdx.x;
	int tid_y = blockIdx.y * blockDim.y + threadIdx.y;
	int tid = tid_x + tid_y;

	if (tid_x < N && tid_y < N)
	{
		//runs Newton's formula
		float dx = bodies[tid_x].x - bodies[tid_y].x;
		float dy = bodies[tid_x].y - bodies[tid_y].y;
		if (tid < 0)
			printf("%f %f", dx, dy);

		float dist = dx * dx + dy * dy;

		float force, dist_inv;
		if ((bodies[tid_x].r + bodies[tid_y].r) * (bodies[tid_x].r + bodies[tid_y].r) >= dist)
		{
			force = 0;
			dist_inv = 0;
		}
		else
		{
			force = -(bodies[tid_x].m * bodies[tid_y].m * g) / dist;
			dist_inv = rsqrtf(dist);  // ignore the squiggle lines, this is a CUDA inbuilt function
		}

		dx *= dist_inv;
		// no writing on the iterator output field
		// a) would not work anyway, because thousands of GPU threads will try to do it simultaneously and corrupt the result and
		// b) we get the correct result anyway, because when the iterator changes the previous iterator's values will be changed then
		//v_x[it] += dx * force * t / m[it];
		bodies[tid_y].v_x -= dx * force * t / bodies[tid_y].m;

		dy *= dist_inv;
		//v_y[it] += dy * force * t / m[it];
		bodies[tid_y].v_y -= dy * force * t / bodies[tid_y].m;
	}

}

__global__ void calc_movement_cuda(body* bodies, float t, int N)
{

	// global thread ID using the formula from the CUDA docs
	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	if (tid < N)
	{
		bodies[tid].x += 0.5 * bodies[tid].v_x * t;
		bodies[tid].y += 0.5 * bodies[tid].v_y * t;
	}	

}

void process_bodies_cuda(std::vector<body>& bodies,	body* d_bodies,	sim_settings& ss)
{

	int n_threads = 32;
	int n_blocks = (bodies.size() + n_threads - 1) / n_threads;

	/*
	// run calculation on the GPU
	for (size_t i = 0; i < bodies.size(); i++)
	{
		calc_force_cuda<<<n_blocks, n_threads>>>(d_bodies, ss.g, ss.timestep, bodies.size(), i);
	}
	*/
	dim3 threads_2d(n_threads, n_threads);
	dim3 blocks_2d(n_blocks, n_blocks);
	calc_force_cuda_full<<<blocks_2d, threads_2d>>> (d_bodies, ss.g, ss.timestep, bodies.size());
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
	}
	// wait for all calculations to finish
	hipDeviceSynchronize();
	// run movement calc on the GPU
	calc_movement_cuda<<<n_blocks, n_threads>>>(d_bodies, ss.timestep, bodies.size());

	// copy data back onto the CPU
	const size_t bytes = sizeof(body) * bodies.size();
	hipMemcpy(bodies.data(), d_bodies, bytes, hipMemcpyDeviceToHost);

}